#include "hip/hip_runtime.h"
#include <iostream>
#include "../../common/book.h"

__global__ void add(int a, int b, int *c) {
  *c = a + b;
}

int main(void) {
  int c;
  int *dev_c; // Pointer to store result computed in device

  HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int))); // Allocate sizeof int to dev_c

  add<<<1,1>>>(2, 7, dev_c); // Call device kernel

  HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost)); // Copy result from device pointer to host integer c
  printf("2 + 7 = %d\n", c);
  hipFree(dev_c); // Free the device memory
}

// The device pointer cannot read or write from memory

/*

  Summary of what can be done with device pointer"=

  1) We can pass pointers allocated with hipMalloc() to functions that execute on the device.
  2) We can use pointers allocated with hipMalloc() to read or write memory from code that executes on the device.
  3) We can pass pointers with hipMalloc() to functions that execute on the host.
  4) We cannot use pointers allocated with hipMalloc() to read or write memory from code that executes on the host.

*/

#include "../../common/book.h"

int main(void) {
  hipDeviceProp_t prop;

  int count;
  HANDLE_ERROR(hipGetDeviceCount(&count));

  for (int i = 0; i < count; i++) {
    HANDLE_ERROR(hipGetDeviceProperties(&prop, i)); // This returns a structure of type hipDeviceProp_t

    printf("--- General Information for device %d ---\n", i+1);
    printf("Name: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);

    printf("Device copy overlap: ");
    if (prop.deviceOverlap)
      printf("Enabled\n");
    else
      printf("Disabled\n");

    printf("Kernel execution timeout: ");
    if (prop.kernelExecTimeoutEnabled)
      printf("Enabled\n");
    else
      printf("Disabled\n");

    printf("\n---Memory Information for device %d ---\n", i+1);
    printf("Total global mem: %zu\n", prop.totalGlobalMem);
    printf("Total constant mem: %zu\n", prop.totalConstMem);
    printf("Max mem pitch: %zu\n", prop.memPitch);
    printf("Texture Alignment: %zu\n", prop.textureAlignment);

    printf("\n--- MP Information for device %d ---\n", i+1);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
    printf("Shared mem per mp: %zu\n", prop.sharedMemPerBlock);
    printf("Registers per mp: %d\n", prop.regsPerBlock);
    printf("Threads in warp: %d\n", prop.warpSize);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("\n");
  }
}

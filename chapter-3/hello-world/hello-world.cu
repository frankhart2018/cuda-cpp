
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(void) {} // __global__ indicates that the function is to be run on device (GPU)

int main(void) {
  kernel<<<1,1>>>(); // <<<1,1>>> are the arguments passed to the host, the arguments to device will be as usual inside ().
  printf("Hello, World!\n");
  return 0;
}

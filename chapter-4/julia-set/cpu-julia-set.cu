#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
  float r;
  float i;

  hipComplex(float a, float b) : r(a), i(b) {}

  float magnitude2(void) {
    return r * r + i * i;
  }

  hipComplex operator*(const hipComplex& a) {
    return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
  }

  hipComplex operator+(const hipComplex& a) {
    return hipComplex(r+a.r, i+a.i);
  }
};

int julia(int x, int y) {
  // Center the complex plane at image center
  const float scale = 1.5;
  float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
  float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

  // Translate pixel coordinate to complex space
  hipComplex c(-0.8, 0.156);
  hipComplex a(jx, jy);

  int i = 0;
  for(i = 0; i < 200; i++) {
    a = a * a + c;
    if (a.magnitude2() > 1000)
      return 0;
  }

  return 1;
}

void kernel(unsigned char* ptr) {
  for (int y = 0; y < DIM; y++) {
    for (int x = 0; x < DIM; x++) {
      int offset = x + y * DIM;

      int juliaValue = julia(x, y); // this method returns 1 if the point is in the set and 0 if it is not in the set
      ptr[offset * 4 + 0] = 255 * juliaValue; // Set color to red if it is in set, otherwise it is black
      ptr[offset * 4 + 1] = 0;
      ptr[offset * 4 + 2] = 0;
      ptr[offset * 4 + 3] = 255;
    }
  }
}

int main(void) {
  CPUBitmap bitmap(DIM, DIM);
  unsigned char *ptr = bitmap.get_ptr();

  kernel(ptr);

  bitmap.display_and_exit();
}

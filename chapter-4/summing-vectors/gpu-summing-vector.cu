#include "hip/hip_runtime.h"
#include "../../common/book.h"

#define N 50000

__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x; // handle the data at this index, x is for the x dimension, blockIdx is of two dimensions
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}

int main(void) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  // allocate the memory on the GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

  // fill the arrays 'a' and 'b' on the CPU
  for (int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  // copy the arrays 'a' and 'b' to the GPU
  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

  // Call CUDA kernel
  add<<<N,1>>>(dev_a, dev_b, dev_c); // N blocks, N copies of kernel running in parallel

  // The collection of parallel blocks is called a grid

  // copy the array 'c' back from the GPU to the CPU
  HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  // display the results
  for (int i = 0; i < N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  // free the memory allocated on the GPU
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
